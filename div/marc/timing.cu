#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include <hip/hip_runtime.h>
#include "ker-division.cu.h"
#include "helper.h"

// function calling division kernel
// and does the timing of the kernel
template<uint32_t M, uint32_t Q>
void gpuDiv (int num_instances){
    
    uint32_t total_work = M * num_instances;

    uint32_t uPrec = (total_work);
    uint32_t vPrec = (uPrec);
    uint32_t* u = randBigInt(uPrec, total_work);
    uint32_t* v = randBigInt(vPrec, total_work);
    uint32_t* quo = (uint32_t*)calloc(total_work, sizeof(uint32_t));
    uint32_t* rem = (uint32_t*)calloc(total_work, sizeof(uint32_t));

    // allocate needed memory onto divice.
    uint32_t *d_u, *d_v, *d_quo, *d_rem;
    hipMalloc((void **)&d_u, total_work * sizeof(uint32_t));
    hipMalloc((void **)&d_v, total_work * sizeof(uint32_t));
    hipMalloc((void **)&d_quo, total_work * sizeof(uint32_t));
    hipMalloc((void **)&d_rem, total_work * sizeof(uint32_t));

    // kernel serialization factor 
    //const uint32_t q = 4; // use 69 on 1650 super 


    // copy data from host to device 
    hipMemcpy(d_u, u, total_work * sizeof(uint32_t), hipMemcpyHostToDevice);
    hipMemcpy(d_v, v, total_work * sizeof(uint32_t), hipMemcpyHostToDevice);

    // dry run to load kernel into hardware 
    // what is the point of adding tuborg out of nowhere? bajer is best?
    {
        divShinv<M, Q><<<num_instances, M/Q, 2 * M * sizeof(uint32_t)>>>(d_u, d_v, d_quo, d_rem, num_instances);
        hipDeviceSynchronize();
        gpuAssert(hipPeekAtLastError());
    }

    // time to time. pray for performance 

    {
        uint64_t time_elappsed;
        struct timeval t_start, t_end, t_diff;
        gettimeofday(&t_start,NULL);

        // why 25 runs? follow the masters example 
        for (int i = 0; i < 25; i++){
            divShinv<M, Q><<<num_instances, M/Q, 2 * M * sizeof(uint32_t)>>>(d_u, d_v, d_quo, d_rem, num_instances);
        }

        hipDeviceSynchronize();

        gettimeofday(&t_end,NULL);
        timeval_subtract(&t_diff, &t_end, &t_start);

        time_elappsed = (t_diff.tv_sec*1e6+t_diff.tv_usec) / 25;

        gpuAssert(hipPeekAtLastError());

        // prop uint8 is enough with our kernel.. cachow 
        double runtime_microsecs = time_elappsed;

        printf(" division of %d-bit Big-Int (Base uint_32): device ran %d problem instances \
in %lu micro-seconds \n", M*sizeof(uint32_t), num_instances, runtime_microsecs);


    }

    hipFree(d_u); hipFree(d_v); hipFree(d_quo); hipFree(d_rem);

    free(u); free(v); free(quo); free(rem);
    
}


void printSlice(uint32_t* u, char name, int i, uint32_t m) {
    int min = i-3 < 0 ? 0 : i-3;
    int max = i+3 > m ? m : i+3;

    printf("%c[%u-%u]: [", name, min, max);
    for (int i = min; i < max; i++) {
        printf("%u, ", u[i]);
    }
    printf("]\n");
}

int main() {
  //  srand(time(NULL));
    const uint32_t num_instances = 4;
    const uint32_t M = 1024;
    const uint32_t Q = 8;

    gpuDiv<M,Q>(num_instances);

    return 0;
}