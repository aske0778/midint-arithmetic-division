#include <stdio.h>
#include <stdint.h>
#include <hip/hip_runtime.h>
#include "ker-division.cu.h"

void prnt(const char *str, uint32_t *u, uint32_t m)
{
    printf("%s: [", str);
    for (int i = 0; i < m; i++)
    {
        printf("%u", u[i]);
        if (i < m - 1)
            printf(", ");
    }
    printf("]\n");
}

int main()
{
    uint32_t u[] = {1234, 5678, 91011, 121314};
    uint32_t v[] = {1234, 5678, 91011, 121314};
    uint32_t res[] = {0, 0, 0, 0};
    uint32_t m = 4;

    uint32_t *d_u, *d_v, *d_res;
    hipMalloc((void **)&d_u, m * sizeof(uint32_t));
    hipMalloc((void **)&d_v, m * sizeof(uint32_t));
    hipMalloc((void **)&d_res, m * sizeof(uint32_t));

    hipMemcpy(d_u, u, m * sizeof(uint32_t), hipMemcpyHostToDevice);
    hipMemcpy(d_v, v, m * sizeof(uint32_t), hipMemcpyHostToDevice);

    dim3 blockDim(32, 32);
    div_shinv<<<1, blockDim>>>(d_u, d_v, d_res, m);

    hipMemcpy(res, d_res, m * sizeof(uint32_t), hipMemcpyDeviceToHost);

    prnt("n", res, 4);
    return 0;
}