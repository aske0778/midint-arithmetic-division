#include <stdio.h>
#include <stdint.h>
#include <hip/hip_runtime.h>
#include "ker-division.cu.h"
#include "helper.h"

int main()
{
    const uint32_t M = 1024;
    const uint32_t Q = 8;

    uint32_t uPrec = (M);
    uint32_t* u = randBigInt(uPrec, M);
    uint32_t* v = randBigInt(uPrec * 0.9, M);
    uint32_t* quo = (uint32_t*)calloc(M, sizeof(uint32_t));
    uint32_t* rem = (uint32_t*)calloc(M, sizeof(uint32_t));

    uint32_t *d_u, *d_v, *d_quo, *d_rem;
    hipMalloc((void **)&d_u, M * sizeof(uint32_t));
    hipMalloc((void **)&d_v, M * sizeof(uint32_t));
    hipMalloc((void **)&d_quo, M * sizeof(uint32_t));
    hipMalloc((void **)&d_rem, M * sizeof(uint32_t));

    hipMemcpy(d_u, u, M * sizeof(uint32_t), hipMemcpyHostToDevice);
    hipMemcpy(d_v, v, M * sizeof(uint32_t), hipMemcpyHostToDevice);

    divShinv<M, Q><<<1, M/Q>>>(d_u, d_v, d_quo, d_rem, 1);
    hipDeviceSynchronize();

    hipMemcpy(quo, d_quo, M * sizeof(uint32_t), hipMemcpyDeviceToHost);
    hipMemcpy(rem, d_rem, M * sizeof(uint32_t), hipMemcpyDeviceToHost);

    uint32_t quo_gmp[M] = {0};
    uint32_t rem_gmp[M] = {0};
    div_gmp(u, v, quo_gmp, rem_gmp, M);

    printf("GMP:\n");
    prnt("quo", quo_gmp, M);
    prnt("rem", rem_gmp, M);
    printf("Cuda:\n");
    prnt("quo", quo, M);
    prnt("rem", rem, M);

    return 0;
}