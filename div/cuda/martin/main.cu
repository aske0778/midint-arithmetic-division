#include <stdio.h>
#include <stdint.h>
#include <hip/hip_runtime.h>
#include "../ker-division.cu.h"
#include "../helpers/helper.h"

int main()
{
    //using Base = U32bits;
  //  using Base = U64bits;
   // using Base = U16bits;
    using Base = U8bits;
    using uint_t = Base::uint_t;
    const uint32_t M = 16;
    const uint32_t Q = 4;

    uint_t u[16] = {4, 2, 2, 2, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};
    uint_t v[16] = {4, 1, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};


    // uint_t u[16] = {0, 1, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};
    // uint_t v[16] = {0, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};

    // uint_t u[16] = {37826, 28157, 12125, 41481, 25946, 5930, 13477, 2530, 9635, 36859, 16311, 28179, 0, 0, 0, 0};
    // uint_t v[16] = {32071, 19796, 21146, 45873, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};

    uint_t quo[M] = {0};
    uint_t rem[M] = {0};

    uint_t *d_u, *d_v, *d_quo, *d_rem;
    hipMalloc((void **)&d_u, M * sizeof(uint_t));
    hipMalloc((void **)&d_v, M * sizeof(uint_t));
    hipMalloc((void **)&d_quo, M * sizeof(uint_t));
    hipMalloc((void **)&d_rem, M * sizeof(uint_t));

    hipMemcpy(d_u, u, M * sizeof(uint_t), hipMemcpyHostToDevice);
    hipMemcpy(d_v, v, M * sizeof(uint_t), hipMemcpyHostToDevice);

    divShinv<Base, M, Q><<<1, M/Q, 2 * M * sizeof(uint_t)>>>(d_u, d_v, d_quo, d_rem);
    hipDeviceSynchronize();

    hipMemcpy(quo, d_quo, M * sizeof(uint_t), hipMemcpyDeviceToHost);
    hipMemcpy(rem, d_rem, M * sizeof(uint_t), hipMemcpyDeviceToHost);

    uint_t quo_gmp[M] = {0};
    uint_t rem_gmp[M] = {0};
    div_gmp(u, v, quo_gmp, rem_gmp, M);

    prnt<uint_t>("u", u, M);
    prnt<uint_t>("v", v, M);

    prnt<uint_t>("quo", quo, M);
    prnt<uint_t>("rem", rem, M);
    prnt<uint_t>("quo_gmp", quo_gmp, M);
    prnt<uint_t>("rem_gmp", rem_gmp, M);

    for (int i = 0; i < M; i++)
    {
        if (quo[i] != quo_gmp[i] || rem[i] != rem_gmp[i])
        {
            printf("INVALID \n");
            break;
        }
    }
    return 0;
}