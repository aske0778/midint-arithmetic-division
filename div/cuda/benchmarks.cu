#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include <sys/time.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include "helpers/helper.h"
#include "ker-division.cu.h"

using namespace std;

#define GPU_RUNS_DIV    5
#define ERR         0.000005

#define WITH_VALIDATION 1

#define Q 32

int timeval_subtract(struct timeval *result, struct timeval *t2, struct timeval *t1)
{
    unsigned int resolution=1000000;
    long int diff = (t2->tv_usec + resolution * t2->tv_sec) - (t1->tv_usec + resolution * t1->tv_sec);
    result->tv_sec = diff / resolution;
    result->tv_usec = diff % resolution;
    return (diff<0);
}

/**
 * Number of giga-u32-bit unit operations.
 */
template<typename uint_t>
uint64_t numAd32OpsOfDivInst(uint32_t m0) {
    uint32_t m = m0*sizeof(uint_t) / 4;
    uint32_t lgm = 0, mm = m;
    for( ; mm > 1; mm >>= 1) lgm++;
    uint64_t fft_cost = 300 * m * lgm;

    uint32_t lgfft = 0, fftc = fft_cost;
    for( ; fftc > 1; fftc >>= 1) lgfft++;
    // return fft_cost;
    return 2*fft_cost + 2*lgfft;
}

/**
 * Validates asb(A - B) < ERR
 */
template<class T>
bool validate(T* A, T* B, const uint64_t sizeAB, const T err){
    for(uint64_t i = 0; i < sizeAB; i++) {
        T curr_err = fabs( (A[i] - B[i]) / max(A[i], B[i]) ); 
        if (curr_err >= err) {
            printf("INVALID RESULT at flat index %llu: %f vs %f\n", i, A[i], B[i]);
            return false;
        }
    }
    printf("VALID RESULT!\n");
    return true;
}

/**
 * Validates exactly A == B
 */
template<class T>
bool validateExact(T* A, T* B, uint64_t sizeAB){
    for(uint64_t i = 0; i < sizeAB; i++) {
        if ( A[i] != B[i] ) {
            printf("INVALID RESULT at flat index %lu: %u vs %u\n", i, A[i], B[i]);
            return false;
        }
    }
    printf("VALID RESULT!\n");
    return true;
}


/****************************/
/***** Single Division ******/
/****************************/

template<class Base, uint32_t m>  // m is the size of the big word in Base::uint_t units
void gpuQuo ( uint32_t num_instances
            , typename Base::uint_t* u
            , typename Base::uint_t* v
            , typename Base::uint_t* h_rs
) {
    using uint_t = typename Base::uint_t;
    //using carry_t= typename Base::carry_t;
    
    uint_t* d_as;
    uint_t* d_bs;
    uint_t* d_rs;
    uint32_t mem_size_nums = num_instances * m * sizeof(uint_t);

    
    // 1. allocate device memory
    hipMalloc((void**)&d_as, mem_size_nums);
    hipMalloc((void**)&d_bs, mem_size_nums);
    hipMalloc((void**)&d_rs, mem_size_nums);
 
    // 2. copy host memory to device
    hipMemcpy(d_as, u, mem_size_nums, hipMemcpyHostToDevice);
    hipMemcpy(d_bs, v, mem_size_nums, hipMemcpyHostToDevice);


    // 3. kernel dimensions
    const uint32_t q = Q; // use 8 for A4500 
    
    dim3 block( m/q, 1, 1 );
    dim3 grid ( num_instances, 1, 1);
    uint32_t sh_mem = 2 * m * sizeof(uint_t);

    if (sh_mem >= 64000) { // maximize the amount of shared memory for the kernel
        hipFuncSetAttribute(reinterpret_cast<const void*>(quoShinv<Base), m, q>, hipFuncAttributeMaxDynamicSharedMemorySize, 98000);
    }    

    // 4. dry run
    {
        quoShinv<Base, m, q><<< grid, block, sh_mem >>>(d_as, d_bs, d_rs);
        hipDeviceSynchronize();
        gpuAssert( hipPeekAtLastError() );
    }
    
    const uint32_t x = Base::bits/32;
    assert( (Base::bits >= 32) && (Base::bits % 32 == 0));
    
    // 5. timing instrumentation
    {
        uint64_t elapsed;
        struct timeval t_start, t_end, t_diff;
        gettimeofday(&t_start, NULL); 
        
        for(int i=0; i<GPU_RUNS_DIV; i++) {
            quoShinv<Base, m, q><<< grid, block, sh_mem >>>(d_as, d_bs, d_rs);
        }
        
        hipDeviceSynchronize();

        gettimeofday(&t_end, NULL);
        timeval_subtract(&t_diff, &t_end, &t_start);
        elapsed = (t_diff.tv_sec*1e6+t_diff.tv_usec) / GPU_RUNS_DIV;

        gpuAssert( hipPeekAtLastError() );

        double runtime_microsecs = elapsed; 
        double num_u32_ops = num_instances * numAd32OpsOfDivInst<uint_t>(m);
        double gigaopsu32 = num_u32_ops / (runtime_microsecs * 1000);

        printf( "Quotient on %d-bit Big-Numbers (base u%d) runs %d instances in: \
%lu microsecs, Gu32ops/sec: %.2f, Mil-Instances/sec: %.2f\n"
              , m*x*32, Base::bits, num_instances, elapsed, gigaopsu32, num_instances / runtime_microsecs
              );
    }
    
    hipMemcpy(h_rs, d_rs, mem_size_nums, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    hipFree(d_as);
    hipFree(d_bs);
    hipFree(d_rs);
}


template<class Base, uint32_t m>  // m is the size of the big word in Base::uint_t units
void gpuDiv ( uint32_t num_instances
            , typename Base::uint_t* u
            , typename Base::uint_t* v
            , typename Base::uint_t* h_quo
            , typename Base::uint_t* h_rem
) {
    using uint_t = typename Base::uint_t;
    //using carry_t= typename Base::carry_t;
    
    uint_t* d_as;
    uint_t* d_bs;
    uint_t* d_quo;
    uint_t* d_rem;
    uint32_t mem_size_nums = num_instances * m * sizeof(uint_t);

    // 1. allocate device memory
    hipMalloc((void**)&d_as, mem_size_nums);
    hipMalloc((void**)&d_bs, mem_size_nums);
    hipMalloc((void**)&d_quo, mem_size_nums);
    hipMalloc((void**)&d_rem, mem_size_nums);
 
    // 2. copy host memory to device
    hipMemcpy(d_as, u, mem_size_nums, hipMemcpyHostToDevice);
    hipMemcpy(d_bs, v, mem_size_nums, hipMemcpyHostToDevice);

    // 3. kernel dimensions
    const uint32_t q = Q; // use 8 for A4500 

    dim3 block( m/q, 1, 1 );
    dim3 grid ( num_instances, 1, 1);
    uint32_t sh_mem = 2 * m * sizeof(uint_t);

    if (sh_mem >= 64000) { // maximize the amount of shared memory for the kernel
        hipFuncSetAttribute(reinterpret_cast<const void*>(divShinv<Base), m, q>, hipFuncAttributeMaxDynamicSharedMemorySize, 98000);
    }    
    
    // 4. dry run
    {
        divShinv<Base, m, q><<< grid, block, sh_mem >>>(d_as, d_bs, d_quo, d_rem);
        hipDeviceSynchronize();
        gpuAssert( hipPeekAtLastError() );
    }
    
    const uint32_t x = Base::bits/32;
    assert( (Base::bits >= 32) && (Base::bits % 32 == 0));
    
    // 5. timing instrumentation
    {
        uint64_t elapsed;
        struct timeval t_start, t_end, t_diff;
        gettimeofday(&t_start, NULL); 
        
        for(int i=0; i<GPU_RUNS_DIV; i++) {
            divShinv<Base, m, q><<< grid, block, sh_mem >>>(d_as, d_bs, d_quo, d_rem);
        }
        
        hipDeviceSynchronize();

        gettimeofday(&t_end, NULL);
        timeval_subtract(&t_diff, &t_end, &t_start);
        elapsed = (t_diff.tv_sec*1e6+t_diff.tv_usec) / GPU_RUNS_DIV;

        gpuAssert( hipPeekAtLastError() );

        double runtime_microsecs = elapsed; 
        double num_u32_ops = num_instances * numAd32OpsOfDivInst<uint_t>(m);
        double gigaopsu32 = num_u32_ops / (runtime_microsecs * 1000);

        printf( "Division on %d-bit Big-Numbers (base u%d) runs %d instances in: \
%lu microsecs, Gu32ops/sec: %.2f, Mil-Instances/sec: %.2f\n"
              , m*x*32, Base::bits, num_instances, elapsed, gigaopsu32, num_instances / runtime_microsecs
              );
    }
    
    hipMemcpy(h_quo, d_quo, mem_size_nums, hipMemcpyDeviceToHost);
    hipMemcpy(h_rem, d_rem, mem_size_nums, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    hipFree(d_as);
    hipFree(d_bs);
    hipFree(d_quo);
    hipFree(d_rem);
}


template<class Base, int m>
void testQuotient( int num_instances
                 , typename Base::uint_t* res_gmp
                 , typename Base::uint_t* res_our
                 , uint32_t with_validation
) {
    using uint_t = typename Base::uint_t;
    
    uint_t uPrec = m;
    uint_t vPrec = uPrec - (m/4);

    uint_t* u = randBigInt<uint_t>(uPrec, m, num_instances);
    uint_t* v = randBigInt<uint_t>(vPrec, m, num_instances);

    const uint32_t x = Base::bits/32;
    assert( (Base::bits >= 32) && (Base::bits % 32 == 0));

    if(with_validation)
        gmpQuo<uint_t, m>(num_instances, u, v, res_gmp);

    gpuQuo<Base, m/x>(num_instances, u, v, res_our);

    if(with_validation)  
        validateExact(res_gmp, res_our, num_instances*m);
}

template<class Base, int m>
void testDivision( int num_instances
                 , typename Base::uint_t* gmp_quo
                 , typename Base::uint_t* gmp_rem
                 , typename Base::uint_t* our_quo
                 , typename Base::uint_t* our_rem
                 , uint32_t with_validation
) {
    using uint_t = typename Base::uint_t;
    
    uint_t uPrec = m;
    uint_t vPrec = uPrec - (m/4);

    uint_t* u = randBigInt<uint_t>(uPrec, m, num_instances);
    uint_t* v = randBigInt<uint_t>(vPrec, m, num_instances);

    const uint32_t x = Base::bits/32;
    assert( (Base::bits >= 32) && (Base::bits % 32 == 0));

    if(with_validation)
        gmpDiv<uint_t, m>(num_instances, u, v, gmp_quo, gmp_rem);

    gpuDiv<Base, m/x>(num_instances, u, v, our_quo, our_rem);

    if(with_validation)  
        validateExact(gmp_quo, our_quo, num_instances*m);
        validateExact(gmp_rem, our_rem, num_instances*m);
}


/////////////////////////////////////////////////////////
// Main program that runs test suits
/////////////////////////////////////////////////////////
 
template<typename Base>
void runQuotients(uint64_t total_work) {

    using uint_t = typename Base::uint_t;
    uint_t *res_gmp, *res_our;

    res_our = (uint_t*)calloc(total_work, sizeof(uint_t));
    res_gmp = (uint_t*)calloc(total_work, sizeof(uint_t));
    
#if 1
    // testQuotient<Base, 8192>( total_work/8192, res_gmp, res_our, WITH_VALIDATION );
    // testQuotient<Base, 4096>( total_work/4096, res_gmp, res_our, WITH_VALIDATION );
    // testQuotient<Base, 2048>( total_work/2048, res_gmp, res_our, WITH_VALIDATION );
    // testQuotient<Base, 1024>( total_work/1024, res_gmp, res_our, WITH_VALIDATION );
    // testQuotient<Base,  512>( total_work/512,  res_gmp, res_our, WITH_VALIDATION );
    // testQuotient<Base,  256>( total_work/256,  res_gmp, res_our, WITH_VALIDATION );
    // testQuotient<Base,  128>( total_work/128,  res_gmp, res_our, WITH_VALIDATION );
    // testQuotient<Base,   64>( total_work/64,   res_gmp, res_our, WITH_VALIDATION );
    testQuotient<Base,   32>( total_work/32,   res_gmp, res_our, WITH_VALIDATION );
    // testQuotient<Base,   16>( total_work/16,   res_gmp, res_our, WITH_VALIDATION );
#endif
    free(res_gmp);
    free(res_our);
}

template<typename Base>
void runDivisions(uint64_t total_work) {

    using uint_t = typename Base::uint_t;
    uint_t *gmp_quo, *gmp_rem, *our_quo, *our_rem;

    gmp_quo = (uint_t*)calloc(total_work, sizeof(uint_t));
    gmp_rem = (uint_t*)calloc(total_work, sizeof(uint_t));
    our_quo = (uint_t*)calloc(total_work, sizeof(uint_t));
    our_rem = (uint_t*)calloc(total_work, sizeof(uint_t));

#if 1
    // testDivision<Base, 8192>( total_work/8192, gmp_quo, gmp_rem, our_quo, our_rem, WITH_VALIDATION );
    // testDivision<Base, 4096>( total_work/4096, gmp_quo, gmp_rem, our_quo, our_rem, WITH_VALIDATION );
    // testDivision<Base, 2048>( total_work/2048, gmp_quo, gmp_rem, our_quo, our_rem, WITH_VALIDATION );
    // testDivision<Base, 1024>( total_work/1024, gmp_quo, gmp_rem, our_quo, our_rem, WITH_VALIDATION );
    // testDivision<Base,  512>( total_work/512,  gmp_quo, gmp_rem, our_quo, our_rem, WITH_VALIDATION );
    // testDivision<Base,  256>( total_work/256,  gmp_quo, gmp_rem, our_quo, our_rem, WITH_VALIDATION );
    // testDivision<Base,  128>( total_work/128,  gmp_quo, gmp_rem, our_quo, our_rem, WITH_VALIDATION );
    // testDivision<Base,   64>( total_work/64,   gmp_quo, gmp_rem, our_quo, our_rem, WITH_VALIDATION );
    testDivision<Base,   32>( total_work/32,   gmp_quo, gmp_rem, our_quo, our_rem, WITH_VALIDATION );
    // testDivision<Base,   16>( total_work/16,   gmp_quo, gmp_rem, our_quo, our_rem, WITH_VALIDATION );
#endif
    free(gmp_quo);
    free(gmp_rem);
    free(our_quo);
    free(our_rem);
}

 
int main (int argc, char * argv[]) {
    if (argc != 2) {
        printf("Usage: %s <batch-size>\n", argv[0]);
        exit(1);
    }
        
    const int total_work = atoi(argv[1]);

    hipSetDevice(1);

    runQuotients<U32bits>(total_work);
    runDivisions<U32bits>(total_work);
}
