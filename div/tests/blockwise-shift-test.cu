#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <stdint.h>
#include "../ker-division.cu.h"
// #include "../sequential/helper.h"
// #include "../sequential/div.h"
#include "../../cuda/helper.h"

__global__ void CallShift(
    const int n,
    const uint32_t* u,
    uint32_t* r,
    const uint32_t m) {
        BlockwiseShift<8>(n, u, r, m);
    }

void printSlice(uint32_t* u, char name, int i, uint32_t m) {
    int min = i-3 < 0 ? 0 : i-3;
    int max = i+3 > m ? m : i+3;

    printf("%c[%d-%d]: [", name, min, max);
    for (int i = min; i < max; i++) {
        printf("%d, ", u[i]);
    }
    printf("]\n");
}

void shift(int n, uint32_t* u, uint32_t* r, uint32_t m)
{
    if (n >= 0)
    { // Right shift
        for (int i = m - 1; i >= 0; i--)
        {
            int offset = i - n;
            r[i] = (offset >= 0) ? u[offset] : 0;
        }
    }
    else
    { // Left shift
        for (int i = 0; i < m; i++)
        {
            int offset = i - n;
            r[i] = (offset < m) ? u[offset] : 0;
        }
    }
}



int main(int argc, char* argv[]) {
    if (argc != 1) {
        printf("Usage-fixd: %s 0 <m> <space-seperated big-ints>\n", argv[0]);
        printf("Usage-rand: %s 1 <m>\n", argv[0]);
        exit(1);
    }


    uint32_t m = 100;
    int size = m * sizeof(uint32_t);

    // uint32_t u[10] = {0, 0, 0, 0, 0, 0, 0, 0, 0, 0};
    uint32_t* u = (uint32_t*)malloc(size);
    uint32_t* v = (uint32_t*)malloc(size);
    uint32_t* v_D;
    hipMalloc(&v_D, size);

    for (int j = 0; j < 10; j++) {
        srand(time(NULL));
        int shiftBy = (rand() % 110) - 10;

        randomInit<uint32_t>(u, m);
        hipMemcpy(v_D, u, size, hipMemcpyHostToDevice);

        shift(shiftBy, u, u, m);

        int threadsPerBlock = 256;
        CallShift<<<1, threadsPerBlock>>>(shiftBy, v_D, v_D, m);
        hipDeviceSynchronize();

        gpuAssert( hipPeekAtLastError() );
        hipMemcpy(v, v_D, size, hipMemcpyDeviceToHost);

        printf("%d\n", shiftBy);
        for (int i = 0; i < m; i++) {
            if (v[i] != u[i]) {
                printf("ERROR AT ITERATION: %d\n", j);
                printSlice(u, 'u', i, m);
                printSlice(v, 'v', i, m);

                printf("INVALID AT INDEX %d: [%d/%d]\n", i, v[i], u[i]);

                // free(u);
                free(v);
                hipFree(v_D);
                return 1;
            }
        }
    }

    // free(u);
    free(v);
    hipFree(v_D);
    printf("shift: VALID\n");
    return 0;
}








